#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipcub/hipcub.hpp>

#include "HeterogeneousCore/CUDAUtilities/interface/allocate_device.h"
#include "HeterogeneousCore/CUDAUtilities/interface/allocate_host.h"
#include "HeterogeneousCore/CUDAUtilities/interface/currentDevice.h"

#include "HeterogeneousCore/CUDAUtilities/interface/host_unique_ptr.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"


#include "clusterGPU.cuh"

#define GPU_DEBUG

using fedId_t = stripgpu::fedId_t;
using fedCh_t = stripgpu::fedCh_t;
using detId_t = stripgpu::detId_t;
using stripId_t = stripgpu::stripId_t;

__global__
static void setSeedStripsGPU(sst_data_t *sst_data_d, const SiStripConditionsGPU * conditions) {
  const int nStrips = sst_data_d->nStrips;
  const uint8_t *__restrict__ adc = sst_data_d->adc;
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
  const fedId_t *__restrict__ fedId = sst_data_d->fedId;
  const fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;
  int *__restrict__ seedStripsMask = sst_data_d->seedStripsMask;
  int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  const float SeedThreshold = 3.0;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    seedStripsMask[i] = 0;
    seedStripsNCMask[i] = 0;
    stripId_t strip = stripId[i];
    if (strip != stripgpu::invStrip) {
      fedId_t fed = fedId[i];
      fedCh_t channel = fedCh[i];
      float noise_i = conditions->noise(fed, channel, strip);
      uint8_t adc_i = adc[i];
      seedStripsMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? 1:0;
      seedStripsNCMask[i] = seedStripsMask[i];
    }
  }
}

__global__
static void setNCSeedStripsGPU(sst_data_t *sst_data_d) {
  const int nStrips = sst_data_d->nStrips;
  const detId_t *__restrict__ detId = sst_data_d->detId;
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
  const int *__restrict__ seedStripsMask = sst_data_d->seedStripsMask;
  int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i>0&&i<nStrips) {
    if (seedStripsMask[i]&&seedStripsMask[i-1]&&(stripId[i]-stripId[i-1])==1&&(detId[i]==detId[i-1])) seedStripsNCMask[i] = 0;
  }
}

__global__
static void setStripIndexGPU(sst_data_t *sst_data_d) {
  const int nStrips = sst_data_d->nStrips;
  const int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;
  const int *__restrict__ prefixSeedStripsNCMask = sst_data_d->prefixSeedStripsNCMask;
  int *__restrict__ seedStripsNCIndex = sst_data_d->seedStripsNCIndex;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    if (seedStripsNCMask[i] == 1) {
      int index = prefixSeedStripsNCMask[i];
      seedStripsNCIndex[index] = i;
    }
  }
}

__global__
static void findLeftRightBoundaryGPU(sst_data_t *sst_data_d, const SiStripConditionsGPU *conditions, clust_data_t *clust_data_d) {
  const int nStrips = sst_data_d->nStrips;
  const int *__restrict__ seedStripsNCIndex = sst_data_d->seedStripsNCIndex;
  const int nSeedStripsNC = sst_data_d->nSeedStripsNC;
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
  const detId_t *__restrict__ detId = sst_data_d->detId;
  const uint8_t *__restrict__ adc = sst_data_d->adc;
  const fedId_t *__restrict__ fedId = sst_data_d->fedId;
  const fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;

  int *__restrict__ clusterLastIndexLeft = clust_data_d->clusterLastIndexLeft;
  int *__restrict__ clusterLastIndexRight = clust_data_d->clusterLastIndexRight;
  bool *__restrict__ trueCluster = clust_data_d->trueCluster;

   const uint8_t MaxSequentialHoles = 0;
   const float  ChannelThreshold = 2.0;
   const float ClusterThresholdSquared = 25.0;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int index, testIndexLeft, testIndexRight, indexLeft, indexRight, rangeLeft, rangeRight;
   uint8_t testADC;
   float noise_i, testNoise, noiseSquared_i, adcSum_i;
   bool noiseSquaredPass, sameDetLeft, sameDetRight;
   fedId_t fed, testFed;
   fedCh_t channel, testChannel;
   stripId_t strip, testStrip;
   int i = nthreads * bid + tid;

   if (i<nSeedStripsNC) {
     index=seedStripsNCIndex[i];
     indexLeft = index;
     indexRight = index;
     fed = fedId[index];
     channel = fedCh[index];
     strip = stripId[index];
     noise_i = conditions->noise(fed, channel, strip);
     noiseSquared_i = noise_i*noise_i;
     adcSum_i = static_cast<float>(adc[index]);

     // find left boundary
     testIndexLeft=index-1;
     if (testIndexLeft>=0) {
       rangeLeft = stripId[indexLeft]-stripId[testIndexLeft]-1;
       sameDetLeft = detId[index] == detId[testIndexLeft];
       while(sameDetLeft&&testIndexLeft>=0&&rangeLeft>=0&&rangeLeft<=MaxSequentialHoles) {
         testFed = fedId[testIndexLeft];
         testChannel = fedCh[testIndexLeft];
         testStrip = stripId[testIndexLeft];
         testNoise = conditions->noise(testFed, testChannel, testStrip);
         testADC = adc[testIndexLeft];

         if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
           --indexLeft;
           noiseSquared_i += testNoise*testNoise;
           adcSum_i += static_cast<float>(testADC);
         }
         --testIndexLeft;
         if (testIndexLeft>=0) {
           rangeLeft = stripId[indexLeft]-stripId[testIndexLeft]-1;
           sameDetLeft = detId[index] == detId[testIndexLeft];
         }
       }
     }

     // find right boundary
     testIndexRight=index+1;
     if (testIndexRight<nStrips) {
       rangeRight = stripId[testIndexRight]-stripId[indexRight]-1;
       sameDetRight = detId[index] == detId[testIndexRight];
       while(sameDetRight&&testIndexRight<nStrips&&rangeRight>=0&&rangeRight<=MaxSequentialHoles) {
         testFed = fedId[testIndexRight];
         testChannel = fedCh[testIndexRight];
         testStrip = stripId[testIndexRight];
         testNoise = conditions->noise(testFed, testChannel, testStrip);
         testADC = adc[testIndexRight];
         if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
           ++indexRight;
           noiseSquared_i += testNoise*testNoise;
           adcSum_i += static_cast<float>(testADC);
         }
         ++testIndexRight;
         if (testIndexRight<nStrips) {
           rangeRight = stripId[testIndexRight]-stripId[indexRight]-1;
           sameDetRight = detId[index] == detId[testIndexRight];
         }
       }
     }
     noiseSquaredPass = noiseSquared_i*ClusterThresholdSquared <= adcSum_i*adcSum_i;
     trueCluster[i] = noiseSquaredPass;
     clusterLastIndexLeft[i] = indexLeft;
     clusterLastIndexRight[i] = indexRight;

   }
}

__global__
static void checkClusterConditionGPU(sst_data_t *sst_data_d, const SiStripConditionsGPU *conditions, clust_data_t *clust_data_d) {
   const uint16_t *__restrict__ stripId = sst_data_d->stripId;
   const uint8_t *__restrict__ adc = sst_data_d->adc;
   const fedId_t *__restrict__ fedId = sst_data_d->fedId;
   const fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;
   const int nSeedStripsNC = sst_data_d->nSeedStripsNC;
   const int *__restrict__ clusterLastIndexLeft = clust_data_d->clusterLastIndexLeft;
   const int *__restrict__ clusterLastIndexRight = clust_data_d->clusterLastIndexRight;
   uint8_t *__restrict__ clusterADCs = clust_data_d->clusterADCs;
   bool *__restrict__ trueCluster = clust_data_d->trueCluster;
   float *__restrict__ barycenter = clust_data_d->barycenter;
   const float minGoodCharge = 1620.0;
   const uint16_t stripIndexMask = 0x7FFF;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   const int i = nthreads * bid + tid;

   int left, right, size, j;
   int charge;
   uint8_t adc_j;
   float gain_j;
   float adcSum=0.0f;
   int sumx=0;
   int suma=0;
   fedId_t fed;
   fedCh_t channel;
   stripId_t strip;

   if (i<nSeedStripsNC) {
     if (trueCluster[i]) {
       left=clusterLastIndexLeft[i];
       right=clusterLastIndexRight[i];
       size=right-left+1;

       if (i>0 && clusterLastIndexLeft[i-1]==left) {
         trueCluster[i] = 0;  // ignore duplicates
       } else {
         for (j=0; j<size; j++){
           adc_j = adc[left+j];
           fed = fedId[left+j];
           channel = fedCh[left+j];
           strip = stripId[left+j];
           gain_j = conditions->gain(fed, channel, strip);
           charge = static_cast<int>( static_cast<float>(adc_j)/gain_j + 0.5f );
           if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
           if (j < kClusterMaxStrips) {
             clusterADCs[j*nSeedStripsNC+i] = adc_j;
           }
           adcSum += static_cast<float>(adc_j);
           sumx += j*adc_j;
           suma += adc_j;
         }
         barycenter[i] = static_cast<float>(stripId[left] & stripIndexMask) + static_cast<float>(sumx)/static_cast<float>(suma) + 0.5f;
       }
       trueCluster[i] = (adcSum*conditions->invthick(fed, channel)) > minGoodCharge;
     }
   }
}

void allocateSSTDataGPU(int max_strips, StripDataGPU& stripdata, sst_data_t *sst_data_d, sst_data_t **pt_sst_data_d, hipStream_t stream) {
  int dev = cms::cuda::currentDevice();
  *pt_sst_data_d = (sst_data_t *)cms::cuda::allocate_device(dev, sizeof(sst_data_t), stream);
  sst_data_d->detId = stripdata.detIdGPU_.get();
  sst_data_d->stripId = stripdata.stripIdGPU_.get();
  sst_data_d->adc = stripdata.alldataGPU_.get();
  sst_data_d->fedId = stripdata.fedIdGPU_.get();
  sst_data_d->fedCh = stripdata.fedChGPU_.get();
  sst_data_d->seedStripsMask = (int *)cms::cuda::allocate_device(dev, 2*max_strips*sizeof(int), stream);
  sst_data_d->prefixSeedStripsNCMask = (int *)cms::cuda::allocate_device(dev, 2*max_strips*sizeof(int), stream);

  sst_data_d->seedStripsNCMask = sst_data_d->seedStripsMask + max_strips;
  sst_data_d->seedStripsNCIndex = sst_data_d->prefixSeedStripsNCMask + max_strips;
  sst_data_d->d_temp_storage=NULL;
  sst_data_d->temp_storage_bytes=0;
  hipcub::DeviceScan::ExclusiveSum(sst_data_d->d_temp_storage, sst_data_d->temp_storage_bytes, sst_data_d->seedStripsNCMask, sst_data_d->prefixSeedStripsNCMask, sst_data_d->nStrips);
#ifdef GPU_DEBUG
  std::cout<<"temp_storage_bytes="<<sst_data_d->temp_storage_bytes<<std::endl;
#endif

  sst_data_d->d_temp_storage = cms::cuda::allocate_device(dev, sst_data_d->temp_storage_bytes, stream);
  cudaCheck(hipMemcpyAsync((void *)*pt_sst_data_d, sst_data_d, sizeof(sst_data_t), hipMemcpyHostToDevice, stream));
}

void allocateClustDataGPU(int max_strips, clust_data_t *clust_data_d, clust_data_t **pt_clust_data_d, hipStream_t stream) {
  int dev = cms::cuda::currentDevice();

  *pt_clust_data_d = (clust_data_t *)cms::cuda::allocate_device(dev, sizeof(clust_data_t), stream);
  clust_data_d->clusterLastIndexLeft = (int *)cms::cuda::allocate_device(dev, 2*max_strips*sizeof(int), stream);
  clust_data_d->clusterADCs = (uint8_t *)cms::cuda::allocate_device(dev, max_strips*kClusterMaxStrips*sizeof(uint8_t), stream);
  clust_data_d->trueCluster = (bool *)cms::cuda::allocate_device(dev, max_strips*sizeof(bool), stream);
  clust_data_d->barycenter = (float *)cms::cuda::allocate_device(dev, max_strips*sizeof(float), stream);
  clust_data_d->clusterLastIndexRight = clust_data_d->clusterLastIndexLeft + max_strips;
  cudaCheck(hipMemcpyAsync((void *)*pt_clust_data_d, clust_data_d, sizeof(clust_data_t), hipMemcpyHostToDevice, stream));
}

void allocateClustData(int max_seedstrips, clust_data_t *clust_data, hipStream_t stream){
  clust_data->clusterLastIndexLeft = (int *)cms::cuda::allocate_host(2*max_seedstrips*sizeof(int), stream);
  clust_data->clusterLastIndexRight = clust_data->clusterLastIndexLeft + max_seedstrips;
  clust_data->clusterADCs = (uint8_t*)cms::cuda::allocate_host(max_seedstrips*kClusterMaxStrips*sizeof(uint8_t), stream);
  clust_data->trueCluster = (bool *)cms::cuda::allocate_host(max_seedstrips*sizeof(bool), stream);
  clust_data->barycenter = (float *)cms::cuda::allocate_host(max_seedstrips*sizeof(float), stream);
}

void freeSSTDataGPU(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, hipStream_t stream) {
  int dev = cms::cuda::currentDevice();
  cms::cuda::free_device(dev, pt_sst_data_d);
  cms::cuda::free_device(dev, sst_data_d->seedStripsMask);
  cms::cuda::free_device(dev, sst_data_d->prefixSeedStripsNCMask);
}

void freeClustDataGPU(clust_data_t *clust_data_d, clust_data_t *pt_clust_data_d, hipStream_t stream) {
  int dev = cms::cuda::currentDevice();
  cms::cuda::free_device(dev, pt_clust_data_d);
  cms::cuda::free_device(dev, clust_data_d->clusterLastIndexLeft);
  cms::cuda::free_device(dev, clust_data_d->clusterADCs);
  cms::cuda::free_device(dev, clust_data_d->trueCluster);
  cms::cuda::free_device(dev, clust_data_d->barycenter);
}

void freeClustData(clust_data_t *clust_data) {
  cms::cuda::free_host(clust_data->clusterLastIndexLeft);
  cms::cuda::free_host(clust_data->clusterADCs);
  cms::cuda::free_host(clust_data->trueCluster);
  cms::cuda::free_host(clust_data->barycenter);
}

void findClusterGPU(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, const SiStripConditionsGPU *conditions, clust_data_t *clust_data_d, clust_data_t *pt_clust_data_d, hipStream_t stream) {
  const int nthreads = 128;
  const int nSeedStripsNC = sst_data_d->nSeedStripsNC;
  const int nblocks = (nSeedStripsNC+nthreads-1)/nthreads;

#ifdef GPU_DEBUG
  int nStrips = sst_data_d->nStrips;
  auto cpu_index = cms::cuda::make_host_unique<int[]>(nStrips, stream);
  auto cpu_strip = cms::cuda::make_host_unique<uint16_t[]>(nStrips, stream);
  auto cpu_adc = cms::cuda::make_host_unique<uint8_t[]>(nStrips, stream);
  auto cpu_noise = cms::cuda::make_host_unique<float[]>(nStrips, stream);

  cudaCheck(hipMemcpyAsync(cpu_strip.get(), sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(cpu_adc.get(), sst_data_d->adc, nStrips*sizeof(uint8_t), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(cpu_index.get(), sst_data_d->seedStripsNCIndex, nStrips*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipStreamSynchronize(stream));

  for (int i=0; i<nStrips; i++) {
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<(unsigned int)cpu_adc[i]<<" cpu index "<<cpu_index[i]<<std::endl;
  }
#endif

  std::cout << "findLeftRightBoundaryGPU sync"; hipStreamSynchronize(stream);
  findLeftRightBoundaryGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d, conditions, pt_clust_data_d);
  cudaCheck(hipGetLastError());

  std::cout << "checkClusterConditionGPU sync"; hipStreamSynchronize(stream);
  checkClusterConditionGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d, conditions, pt_clust_data_d);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  auto clusterLastIndexLeft = cms::cuda::make_host_unique<int[]>(nSeedStripsNC, stream);
  auto clusterLastIndexRight = cms::cuda::make_host_unique<int[]>(nSeedStripsNC, stream);
  auto trueCluster = cms::cuda::make_host_unique<bool[]>(nSeedStripsNC, stream);
  auto ADCs = cms::cuda::make_host_unique<uint8_t[]>(nSeedStripsNC*kClusterMaxStrips, stream);

  std::cout<< "findClusterGPU sync"; hipStreamSynchronize(stream);

  std::cout<<"findClusterGPU nSeedStripsNC="<<nSeedStripsNC<<std::endl;
  cudaCheck(hipMemcpyAsync(clusterLastIndexLeft.get(), clust_data_d->clusterLastIndexLeft, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost, 
stream));
  cudaCheck(hipMemcpyAsync(clusterLastIndexRight.get(), clust_data_d->clusterLastIndexRight, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(trueCluster.get(), clust_data_d->trueCluster, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(ADCs.get(), clust_data_d->clusterADCs, nSeedStripsNC*kClusterMaxStrips*sizeof(uint8_t), hipMemcpyDeviceToHost, stream));

  hipStreamSynchronize(stream);

  for (int i=0; i<nSeedStripsNC; i++) {
    if (trueCluster[i]){
      int left=clusterLastIndexLeft[i];
      int right=clusterLastIndexRight[i];
      std::cout<<"i="<<i<<" left "<<left<<" right "<<right<<" : ";
      int size=std::min(right-left+1, kClusterMaxStrips);
      for (int j=0; j<size; j++){
        std::cout<<(unsigned int)ADCs[j*nSeedStripsNC+i]<<" ";
      }
      std::cout<<std::endl;
    }
  }

#endif

}

void setSeedStripsNCIndexGPU(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, const SiStripConditionsGPU *conditions, hipStream_t stream) {
#ifdef GPU_DEBUG
  int nStrips = sst_data_d->nStrips;
  auto cpu_strip = cms::cuda::make_host_unique<uint16_t[]>(nStrips, stream);
  auto cpu_adc = cms::cuda::make_host_unique<uint8_t[]>(nStrips, stream);

  cudaCheck(hipMemcpyAsync(cpu_strip.get(), sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(cpu_adc.get(), sst_data_d->adc, nStrips*sizeof(uint8_t), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipStreamSynchronize(stream));

  for (int i=0; i<nStrips; i++) {
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<(unsigned int)cpu_adc[i]<<std::endl;
  }
#endif
  int nthreads = 256;
  int nblocks = (sst_data_d->nStrips+nthreads-1)/nthreads;

  //mark seed strips
  setSeedStripsGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d, conditions);
  cudaCheck(hipGetLastError());

  //mark only non-consecutive seed strips (mask out consecutive seed strips)
  setNCSeedStripsGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d);
  cudaCheck(hipGetLastError());

  hipcub::DeviceScan::ExclusiveSum(sst_data_d->d_temp_storage, sst_data_d->temp_storage_bytes, sst_data_d->seedStripsNCMask, sst_data_d->prefixSeedStripsNCMask, sst_data_d->nStrips, stream);

  cudaCheck(hipMemcpyAsync((void *)&(sst_data_d->nSeedStripsNC), sst_data_d->prefixSeedStripsNCMask+sst_data_d->nStrips-1, sizeof(int), hipMemcpyDeviceToHost, stream));
  hipStreamSynchronize(stream);
  if (sst_data_d->nSeedStripsNC > MAX_SEEDSTRIPS) {
    printf("Seed strips %d greater than max\n", sst_data_d->nSeedStripsNC);
    sst_data_d->nSeedStripsNC = MAX_SEEDSTRIPS;
  }
  cudaCheck(hipMemcpyAsync((void *)&(pt_sst_data_d->nSeedStripsNC), &(sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyHostToDevice, stream));

  setStripIndexGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  auto cpu_mask  = cms::cuda::make_host_unique<int[]>(nStrips, stream);
  auto cpu_prefix= cms::cuda::make_host_unique<int[]>(nStrips, stream);
  auto cpu_index = cms::cuda::make_host_unique<int[]>(nStrips, stream);

  cudaCheck(hipMemcpyAsync(cpu_mask.get(), sst_data_d->seedStripsNCMask, nStrips*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(cpu_prefix.get(), sst_data_d->prefixSeedStripsNCMask, nStrips*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(cpu_index.get(), sst_data_d->seedStripsNCIndex, nStrips*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync(&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipStreamSynchronize(stream));

  std::cout<<"nStrips="<<nStrips<<" nSeedStripsNC="<<sst_data_d->nSeedStripsNC<<" temp_storage_bytes="<<sst_data_d->temp_storage_bytes<<std::endl;
  for (int i=0; i<nStrips; i++) {
    std::cout<<" i "<<i<<" mask "<<cpu_mask[i]<<" prefix "<<cpu_prefix[i]<<" index "<<cpu_index[i]<<std::endl;
  }
#endif
}


void cpyGPUToCPU(sst_data_t * sst_data_d, sst_data_t *pt_sst_data_d, clust_data_t *clust_data, clust_data_t *clust_data_d, hipStream_t stream) {
  const int nSeedStripsNC = sst_data_d->nSeedStripsNC;

  cudaCheck(hipMemcpyAsync((void *)(clust_data->clusterLastIndexLeft), clust_data_d->clusterLastIndexLeft, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync((void *)(clust_data->clusterLastIndexRight), clust_data_d->clusterLastIndexRight, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync((void *)(clust_data->clusterADCs), clust_data_d->clusterADCs, nSeedStripsNC*kClusterMaxStrips*sizeof(uint8_t), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync((void *)(clust_data->trueCluster), clust_data_d->trueCluster, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost, stream));
  cudaCheck(hipMemcpyAsync((void *)(clust_data->barycenter), clust_data_d->barycenter, nSeedStripsNC*sizeof(float), hipMemcpyDeviceToHost, stream));
}
