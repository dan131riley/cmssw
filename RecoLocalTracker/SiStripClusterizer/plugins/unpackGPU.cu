#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

#include "SiStripRawToClusterGPUKernel.h"
#include "ChanLocsGPU.h"
//#include "unpackGPU.cuh"

namespace stripgpu {
  __global__
  static void unpackChannels(const ChanLocStruct* chanlocs, const SiStripConditionsGPU* conditions,
                             uint8_t* alldata, stripgpu::detId_t* detId, stripgpu::stripId_t* stripId,
                             stripgpu::fedId_t* fedId, stripgpu::fedCh_t* fedCh)
  {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int nthreads = blockDim.x;

    const auto chan = nthreads*bid + tid;
    if (chan < chanlocs->size()) {
      const auto fedid = chanlocs->fedID(chan);
      const auto fedch = chanlocs->fedCh(chan);
      const auto detid = conditions->detID(fedid, fedch);
      const auto ipair = conditions->iPair(fedid, fedch);
      const auto ipoff = kStripsPerChannel*ipair;

      const auto data = chanlocs->input(chan);
      const auto len = chanlocs->length(chan);

      if (data != nullptr && len > 0) {
        auto aoff = chanlocs->offset(chan);
        auto choff = chanlocs->inoff(chan);
        const auto end = choff + len;

        while (choff < end) {
          auto stripIndex = data[(choff++)^7] + ipoff;
          const auto groupLength = data[(choff++)^7];

          for (auto i = 0; i < 2; ++i) {
            detId[aoff] = detid;
            fedId[aoff] = fedid;
            fedCh[aoff] = fedch;
            stripId[aoff] = stripgpu::invStrip;
            alldata[aoff++] = 0;
          }

          for (auto i = 0; i < groupLength; ++i) {
            detId[aoff] = detid;
            fedId[aoff] = fedid;
            fedCh[aoff] = fedch;
            stripId[aoff] = stripIndex++;
            alldata[aoff++] = data[(choff++)^7];
          }
        }
      }
    }
  }

  StripDataGPU::StripDataGPU(size_t size, hipStream_t stream)
  {
    alldataGPU_ = cms::cuda::make_device_unique<uint8_t[]>(size, stream);
    detIdGPU_ = cms::cuda::make_device_unique<stripgpu::detId_t[]>(size, stream);
    stripIdGPU_ = cms::cuda::make_device_unique<stripgpu::stripId_t[]>(size, stream);
    fedIdGPU_ = cms::cuda::make_device_unique<stripgpu::fedId_t[]>(size, stream);
    fedChGPU_ = cms::cuda::make_device_unique<stripgpu::fedCh_t[]>(size, stream);
  }

  void SiStripRawToClusterGPUKernel::unpackChannelsGPU(const SiStripConditionsGPU* conditions, hipStream_t stream)
  {
    constexpr int nthreads = 128;
    const auto channels = chanlocsGPU->size();
    const auto nblocks = (channels + nthreads - 1)/nthreads;
  
    unpackChannels<<<nblocks, nthreads, 0, stream>>>(chanlocsGPU->chanLocStruct(), conditions,
                                                     stripdata->alldataGPU_.get(),
                                                     stripdata->detIdGPU_.get(),
                                                     stripdata->stripIdGPU_.get(),
                                                     stripdata->fedIdGPU_.get(),
                                                     stripdata->fedChGPU_.get());
  }
}
